#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <chrono>


using namespace std;

constexpr unsigned int MAX_ITER = 10000;
constexpr unsigned int W_PIX = 65536;
constexpr unsigned int H_PIX = 65536;
constexpr unsigned int X_PIX_PER_BLK = 32;
const string DATA_FILE_NAME = "out.dat";

constexpr unsigned int H_PIX_HALF = H_PIX / 2;

__device__ inline bool run_iter(const double x0, const double y0)
{
    double x = x0;
    double y = y0;
    for (int i = 0; i < MAX_ITER; i++)
    {
        const double x2 = x * x;
        const double y2 = y * y;
        const double hyp2 = x2 + y2;
        if (i == 0 && hyp2 < 1.0 / 16)
            return true;
        if (hyp2 > 4)
            return false;
        y = 2 * x * y + y0;
        x = x2 - y2 + x0;
    }
    return true;
}

__global__ void iter_kernel(unsigned char *device_mem, double x_min, double w, double y_min, double h)
{
    const int pix_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (pix_y > H_PIX_HALF)
        return;
    const int pix_x = threadIdx.x + blockIdx.x * blockDim.x;
    const double x = pix_x * w / W_PIX + x_min;
    const double y = pix_y * h / H_PIX + y_min;
    const int pix_value = run_iter(x, y) ? 0 : 255;
    device_mem[pix_y * W_PIX + pix_x] = static_cast<unsigned char>(pix_value);
}

int main()
{
    int device_cnt;
    hipGetDeviceCount(&device_cnt);
    if (device_cnt <= 0)
    {
        std::cout << "No GPU detected " << endl;
        return 0;
    }
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const int max_parallel_thread = prop.multiProcessorCount * min(prop.maxThreadsPerBlock * prop.maxBlocksPerMultiProcessor, prop.maxThreadsPerMultiProcessor);
    cout << "GPU: \"" << prop.name << "\"\n"
         << " Max threads per block: " << prop.maxThreadsPerBlock << '\n'
         << " Max blocks per SM: " << prop.maxBlocksPerMultiProcessor << '\n'
         << " Max threads per SM: " << prop.maxThreadsPerMultiProcessor << '\n'
         << " Number of SMs: " << prop.multiProcessorCount << '\n'
         << " Max Parallel Threads on Device: " << max_parallel_thread << '\n';
    

    ofstream file("out.dat", ios::out | ios::binary);
    if (!file) {
        cerr << "Cannot open " << DATA_FILE_NAME << endl;
        return 1;
    }
    

    const int y_pix_per_blk = prop.maxThreadsPerBlock / X_PIX_PER_BLK;
    const dim3 blk_dim(X_PIX_PER_BLK, y_pix_per_blk);
    const int y_grid_dim = (H_PIX / y_pix_per_blk) / 2 + 1;
    const dim3 grid_dim(W_PIX / X_PIX_PER_BLK, y_grid_dim);
    unsigned char *mem_device;
    const unsigned int n_pix = W_PIX * (H_PIX_HALF + 1);
    const size_t size_of_mem = n_pix * sizeof(unsigned char);
    hipMalloc((void **)&mem_device, size_of_mem);
    
    auto start = chrono::high_resolution_clock::now();
    iter_kernel<<<grid_dim, blk_dim>>>(mem_device, -2, 4, -2, 4);
    hipDeviceSynchronize();
    auto stop = chrono::high_resolution_clock::now();
    cout << "GPU Time: " << chrono::duration_cast<chrono::microseconds>(stop - start).count() / 1e6 << " seconds" << endl;
    
    unsigned char *mem_host = new unsigned char[n_pix];
    hipMemcpy(mem_host, mem_device, size_of_mem, hipMemcpyDeviceToHost);
    hipFree(mem_device);

    file.write(reinterpret_cast<const char*>(mem_host), size_of_mem);
    delete[] mem_host;
    file.close();
    cout << "Data written to " << DATA_FILE_NAME << endl;
    return 0;
}
